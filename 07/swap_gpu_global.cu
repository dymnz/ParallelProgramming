#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


#include "./common/book.h"

__global__ void swap( int *a, int *b ) {
	int temp = *a;
	*a = *b;
	*b = temp;
}

int main( void ) {
	int a, b;
	int *dev_a, *dev_b;

	a = 7;
	b = 3;
	printf("a:%d b:%d\n", a, b);
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, sizeof(int) ) );
	hipMemcpy(dev_a, (void **) &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, (void **) &b, sizeof(int), hipMemcpyHostToDevice);

	printf("swapping\n");
	swap<<<1,1>>>(dev_a, dev_b);

	HANDLE_ERROR( hipMemcpy( &a, dev_a, sizeof(int),
	                          hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy( &b, dev_b, sizeof(int),
							  hipMemcpyDeviceToHost ) );

	printf("a:%d b:%d\n", a, b);
	HANDLE_ERROR( hipFree(dev_a) );
	HANDLE_ERROR( hipFree(dev_b) );


	return 0;
}


