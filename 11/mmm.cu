#include<hip/hip_runtime.h>
#include<stdio.h>

int main(void) {
	void HostMMM(float *, float *, float *, int);
	void CudaMMM(float *, float *, float *, int);
	const int Width = 5;
	float M[Width*Width], N[Width*Width], P[Width*Width];
	for(int i = 0; i < (Width*Width) ; i++) {
		M[i] = 5;
		N[i] = 5;
		P[i] = 0;
	}

	//HostMMM(M, N, P, Width);
	CudaMMM(M, N, P, Width);

  printf("\n");
	for(int i = 0; i < (Width*Width) ; i++) {
		printf("%f ", P[i]);
		if (i%Width==(Width-1)) printf("\n");
	}
  printf("\n");

	return 0;
}

//HostMMM
void HostMMM(float* A, float* B,  float* C, int n) {
	for (int i = 0; i < n; ++i)
		for (int j = 0; j < n; ++j) {
			double sum = 0;
			for (int k = 0; k < n; ++k) {
				double a = A[i * n + k];
				double b = B[k * n + j];
				sum += a * b;
			}
			C[i * n + j] = sum;
		}
}


//MMMKernel
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
	//2D Thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores the Pd element that is computed by the thread
	float Pvalue = 0;

	for(int k = 0; k < Width ; ++k) {
		float Mdelement = Md[ty*Width + k];
		float Ndelement = Nd[k*Width + tx];
		Pvalue += (Mdelement*Ndelement);
	}

	Pd[ty*Width + tx] = Pvalue;
}

void CudaMMM(float *M, float *N, float *P, int Width) {
	int size = Width*Width*sizeof(float);
	float *Md, *Nd, *Pd;

	//Transfer M and N to device memory
	hipMalloc((void**)&Md, size);
	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

	//Allocate P on the device
	hipMalloc((void**)&Pd,size);

	//Setup the execution configuration
	dim3 dimBlock(Width,Width);
	dim3 dimGrid(1,1);

	//Launch the device computation threads!
	MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

	//Transfer P from device to host
	hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

	//Free device matrices
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
}
